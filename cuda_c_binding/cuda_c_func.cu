
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void my_gpu_func(int* buf, int w, int h) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < w && y < h) {
    buf[y * w + x] += 1;
  }
}

extern "C"{

void my_c_func(int *buf, int *wptr, int *hptr)
{
  int h = *hptr;
  int w = *wptr;

  printf("print from c, w=%d, h=%d\n", w, h);
  for (int y = 0; y < h; ++y) {
    for (int x = 0; x < w; ++x) { 
      printf("%d, ", buf[ y * w + x ]);
    }
    printf("\n");
  }
  printf("\n");

  int *dev_buf;
  size_t size = w * h * sizeof(int);
  hipMalloc((void**)&dev_buf, size);
  hipMemcpy(dev_buf, buf, size, hipMemcpyHostToDevice);

  my_gpu_func<<<1, dim3(w, h)>>>(dev_buf, w, h);

  hipMemcpy(buf, dev_buf, size, hipMemcpyDeviceToHost);
  hipFree(dev_buf);
  return;
}
}
